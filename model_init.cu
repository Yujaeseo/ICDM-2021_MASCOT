#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <chrono>
#include "common_struct.h"
#include "common.h"
#include "model_init.h"
#include <iostream>
using namespace std;

__global__ void init_rand_state(hiprandState*state, int size)
{
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if(tid < size)hiprand_init(clock() + tid, tid, 0, &state[tid]);
    // if(tid < size)hiprand_init(1 + tid, tid, 0, &state[tid]);
    // if(tid < size)hiprand_init(1, tid, 0, &state[tid]);

}

__global__ void init_rand_feature_single(hiprandState*state, unsigned int state_size, float* array , unsigned int dim, unsigned int k)
{
    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int number_threads = gridDim.x*blockDim.x;
    if (state_size > tid){
        for (unsigned int i = tid; i < dim * k; i+= state_size){
            array[i] = (float)((hiprand_normal_double(&state[tid])* 0.01)) ;
            // array[i] = (float)((0.05)) ;
        }
    }
}

__global__ void cpyparams2grouped_params(float* original_params, __half** converted_params, Index_info_node* index_info, unsigned int k, unsigned int n){
    unsigned int g_wid = (blockIdx.x*blockDim.x + threadIdx.x)/32;
    unsigned int num_w = gridDim.x*blockDim.x/32;
    unsigned int lane_id = threadIdx.x%32;

    for (; g_wid < n; g_wid += num_w){
        unsigned int group_idx = index_info[g_wid].g;
        unsigned int base_idx= index_info[g_wid].v * k;
        
        ((__half*)converted_params[group_idx])[base_idx + lane_id] = __float2half_rn(original_params[g_wid * k + lane_id]); 
        ((__half*)converted_params[group_idx])[base_idx + lane_id + 32] = __float2half_rn(original_params[g_wid * k + lane_id + 32]); 
        ((__half*)converted_params[group_idx])[base_idx + lane_id + 64] = __float2half_rn(original_params[g_wid * k + lane_id + 64]); 
        ((__half*)converted_params[group_idx])[base_idx + lane_id + 96] = __float2half_rn(original_params[g_wid * k + lane_id + 96]); 
    }
}

__global__ void cpyparams2grouped_params_fp32_version(float* original_params, float** converted_params, Index_info_node* index_info, unsigned int k, unsigned int n){
    unsigned int g_wid = (blockIdx.x*blockDim.x + threadIdx.x)/32;
    unsigned int num_w = gridDim.x*blockDim.x/32;
    unsigned int lane_id = threadIdx.x%32;

    for (; g_wid < n; g_wid += num_w){
        unsigned int group_idx = index_info[g_wid].g;
        unsigned int base_idx= index_info[g_wid].v * k;
        
        ((float*)converted_params[group_idx])[base_idx + lane_id] = original_params[g_wid * k + lane_id]; 
        ((float*)converted_params[group_idx])[base_idx + lane_id + 32] = original_params[g_wid * k + lane_id + 32]; 
        ((float*)converted_params[group_idx])[base_idx + lane_id + 64] = original_params[g_wid * k + lane_id + 64]; 
        ((float*)converted_params[group_idx])[base_idx + lane_id + 96] = original_params[g_wid * k + lane_id + 96]; 
    }
}


__global__ void cpyparams2grouped_params_for_comparison_indexing(float* original_params, __half** converted_params, unsigned int *group_end_idx, unsigned int *entity2group, unsigned int *entity2sorted_idx, unsigned int k, unsigned int n, unsigned int group_num){
    
    extern __shared__ unsigned int end_idx_s[];

    // if (threadIdx.x < group_num){ end_idx_s[threadIdx.x+1] = group_end_idx[threadIdx.x];}
    for (int i = threadIdx.x; i < group_num; i+= blockDim.x){
        end_idx_s[i+1] = group_end_idx[i];
    }

    if (threadIdx.x == 0){
        end_idx_s[0] = -1;
    }
    __syncthreads();

    unsigned int g_wid = (blockIdx.x*blockDim.x + threadIdx.x)/32;
    unsigned int num_w = gridDim.x*blockDim.x/32;
    unsigned int lane_id = threadIdx.x%32;

    for (; g_wid < n; g_wid += num_w){
        unsigned int group_idx = entity2group[g_wid];
        unsigned int base_idx = (entity2sorted_idx[g_wid]- (end_idx_s[group_idx] + 1)) * k;
        
        ((__half*)converted_params[group_idx])[base_idx + lane_id] = __float2half_rn(original_params[g_wid * k + lane_id]); 
        ((__half*)converted_params[group_idx])[base_idx + lane_id + 32] = __float2half_rn(original_params[g_wid * k + lane_id + 32]); 
        ((__half*)converted_params[group_idx])[base_idx + lane_id + 64] = __float2half_rn(original_params[g_wid * k + lane_id + 64]); 
        ((__half*)converted_params[group_idx])[base_idx + lane_id + 96] = __float2half_rn(original_params[g_wid * k + lane_id + 96]); 
    }
}

__global__ void cpyparams2grouped_params_for_comparison_indexing_k64(float* original_params, __half** converted_params, unsigned int *group_end_idx, unsigned int *entity2group, unsigned int *entity2sorted_idx, unsigned int k, unsigned int n, unsigned int group_num){
    
    extern __shared__ unsigned int end_idx_s[];

    // if (threadIdx.x < group_num){ end_idx_s[threadIdx.x+1] = group_end_idx[threadIdx.x];}
    for (int i = threadIdx.x; i < group_num; i+= blockDim.x){
        end_idx_s[i+1] = group_end_idx[i];
    }

    if (threadIdx.x == 0){
        end_idx_s[0] = -1;
    }
    __syncthreads();

    unsigned int g_wid = (blockIdx.x*blockDim.x + threadIdx.x)/32;
    unsigned int num_w = gridDim.x*blockDim.x/32;
    unsigned int lane_id = threadIdx.x%32;

    for (; g_wid < n; g_wid += num_w){
        unsigned int group_idx = entity2group[g_wid];
        unsigned int base_idx = (entity2sorted_idx[g_wid]- (end_idx_s[group_idx] + 1)) * k;
        
        ((__half*)converted_params[group_idx])[base_idx + lane_id] = __float2half_rn(original_params[g_wid * k + lane_id]); 
        ((__half*)converted_params[group_idx])[base_idx + lane_id + 32] = __float2half_rn(original_params[g_wid * k + lane_id + 32]); 
    }
}

__global__ void cpyparams2grouped_params_for_division_indexing(float* original_params, __half** converted_params, unsigned int* entity2sorted_idx, unsigned int k, unsigned int n, unsigned int group_size){
    
    unsigned int g_wid = (blockIdx.x*blockDim.x + threadIdx.x)/32;
    unsigned int num_w = gridDim.x*blockDim.x/32;
    unsigned int lane_id = threadIdx.x%32;

    for (; g_wid < n; g_wid += num_w){
        unsigned int converted_idx = entity2sorted_idx[g_wid];
        unsigned int group_idx = (converted_idx / group_size);
        unsigned int base_idx= (converted_idx % group_size) * k;
        
        ((__half*)converted_params[group_idx])[base_idx + lane_id] = __float2half_rn(original_params[g_wid * k + lane_id]); 
        ((__half*)converted_params[group_idx])[base_idx + lane_id + 32] = __float2half_rn(original_params[g_wid * k + lane_id + 32]); 
        ((__half*)converted_params[group_idx])[base_idx + lane_id + 64] = __float2half_rn(original_params[g_wid * k + lane_id + 64]); 
        ((__half*)converted_params[group_idx])[base_idx + lane_id + 96] = __float2half_rn(original_params[g_wid * k + lane_id + 96]); 
    }
}

__global__ void cpyparams2grouped_params_for_division_indexing_fp32_version(float* original_params, float** converted_params, unsigned int* entity2sorted_idx, unsigned int k, unsigned int n, unsigned int group_size){
    
    unsigned int g_wid = (blockIdx.x*blockDim.x + threadIdx.x)/32;
    unsigned int num_w = gridDim.x*blockDim.x/32;
    unsigned int lane_id = threadIdx.x%32;

    for (; g_wid < n; g_wid += num_w){
        unsigned int converted_idx = entity2sorted_idx[g_wid];
        unsigned int group_idx = (converted_idx / group_size);
        unsigned int base_idx= (converted_idx % group_size) * k;
        
        ((float*)converted_params[group_idx])[base_idx + lane_id] = __float2half_rn(original_params[g_wid * k + lane_id]); 
        ((float*)converted_params[group_idx])[base_idx + lane_id + 32] = __float2half_rn(original_params[g_wid * k + lane_id + 32]); 
        ((float*)converted_params[group_idx])[base_idx + lane_id + 64] = __float2half_rn(original_params[g_wid * k + lane_id + 64]); 
        ((float*)converted_params[group_idx])[base_idx + lane_id + 96] = __float2half_rn(original_params[g_wid * k + lane_id + 96]); 
    }
}

__global__ void transform_half2float(float *gpu_float_feature, half *gpu_half_feature, unsigned int vec_size)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int number_threads = gridDim.x*blockDim.x;

    for (unsigned int i = tid; i < vec_size; i += number_threads){
        if (i < vec_size)
            gpu_float_feature[i] = __half2float(gpu_half_feature[i]);
    }

}
__global__ void transform_float2half(__half* half_feature, float *gpu_float_feature, unsigned int vec_size){
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int number_threads = gridDim.x*blockDim.x;

    for (unsigned int i = tid; i < vec_size; i += number_threads){
        if (i < vec_size){
            half_feature[i] = __float2half_rn(gpu_float_feature[i]);
        }
    }
}

__global__ void mem_cpy_fp16tofp32(float* out, __half* in, int n){
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    for (; i < n; i += gridDim.x * blockDim.x)
        out[i] = __half2float(in[i]);
}

void init_features_single(float *feature_vec, unsigned int dim, unsigned int k){
    float* gpu_vec;
    hipMalloc(&gpu_vec, sizeof(float) * dim * k);

    unsigned int workers = 3200;
    hiprandState* d_state;
    int state_size = workers * 32;
    
    hipMalloc(&d_state, sizeof(hiprandState) * state_size);
    init_rand_state<<<(state_size + 255)/256, 256>>>(d_state, state_size);
    hipDeviceSynchronize();

    gpuErr(hipPeekAtLastError());

    init_rand_feature_single<<<(state_size + 255)/256, 256>>>(d_state, state_size, gpu_vec, dim, k);
    hipDeviceSynchronize();

    hipMemcpy(feature_vec, gpu_vec, sizeof(float)*dim*k, hipMemcpyDeviceToHost);
    gpuErr(hipPeekAtLastError());
    
    hipFree(d_state);
    hipFree(gpu_vec);
}

void init_features_single_on_device(float *d_feature_vec, unsigned int dim, unsigned int k){
    hipMalloc(&d_feature_vec, sizeof(float) * dim * k);

    unsigned int workers = 3200;
    hiprandState* d_state;
    int state_size = workers * 32;
    
    hipMalloc(&d_state, sizeof(hiprandState) * state_size);
    init_rand_state<<<(state_size + 255)/256, 256>>>(d_state, state_size);
    hipDeviceSynchronize();

    gpuErr(hipPeekAtLastError());

    init_rand_feature_single<<<(state_size + 255)/256, 256>>>(d_state, state_size, d_feature_vec, dim, k);
    hipDeviceSynchronize();
    gpuErr(hipPeekAtLastError());
}

void init_model_single(Mf_info *mf_info, SGD *sgd_info){
    hipHostMalloc(&sgd_info->p, sizeof(float) * mf_info->max_user * mf_info->params.k);
    hipHostMalloc(&sgd_info->q, sizeof(float) * mf_info->max_item * mf_info->params.k);
    gpuErr(hipPeekAtLastError());

    init_features_single(sgd_info->p, mf_info->max_user, mf_info->params.k);
    init_features_single(sgd_info->q, mf_info->max_item, mf_info->params.k);

    hipMalloc(&sgd_info->d_p, sizeof(float) * mf_info->max_user * mf_info->params.k);
    hipMalloc(&sgd_info->d_q, sizeof(float) * mf_info->max_item * mf_info->params.k);

    hipMemcpy(sgd_info->d_p, sgd_info->p, sizeof(float) * mf_info->max_user * mf_info->params.k, hipMemcpyHostToDevice);
    hipMemcpy(sgd_info->d_q, sgd_info->q, sizeof(float) * mf_info->max_item * mf_info->params.k, hipMemcpyHostToDevice);
}

void init_model_half(Mf_info *mf_info, SGD *sgd_info){
    hipHostMalloc(&sgd_info->p, sizeof(float) * mf_info->max_user * mf_info->params.k);
    hipHostMalloc(&sgd_info->q, sizeof(float) * mf_info->max_item * mf_info->params.k);
    gpuErr(hipPeekAtLastError());

    hipHostMalloc(&sgd_info->half_p, sizeof(short) * mf_info->max_user * mf_info->params.k);
    hipHostMalloc(&sgd_info->half_q, sizeof(short) * mf_info->max_item * mf_info->params.k);
    gpuErr(hipPeekAtLastError());

    init_features_single(sgd_info->p, mf_info->max_user, mf_info->params.k);
    init_features_single(sgd_info->q, mf_info->max_item, mf_info->params.k);

    conversion_features_half(sgd_info->half_p, sgd_info->p ,mf_info->max_user, mf_info->params.k);
    conversion_features_half(sgd_info->half_q, sgd_info->q ,mf_info->max_item, mf_info->params.k);

    hipMalloc(&sgd_info->d_half_p, sizeof(short) * mf_info->max_user * mf_info->params.k);
    hipMalloc(&sgd_info->d_half_q, sizeof(short) * mf_info->max_item * mf_info->params.k);

    hipMemcpy(sgd_info->d_half_p, sgd_info->half_p, sizeof(short) * mf_info->max_user * mf_info->params.k, hipMemcpyHostToDevice);
    hipMemcpy(sgd_info->d_half_q, sgd_info->half_q, sizeof(short) * mf_info->max_item * mf_info->params.k, hipMemcpyHostToDevice);
}

void cpy2grouped_parameters(Mf_info *mf_info, SGD *sgd_info){
    //! HOST에서 COPY하지 않고 GPU에서 바로 실행하도록 코드 수정하기
    double cpy2grouped_parameters_exec_time = 0;
    std::chrono::time_point<std::chrono::system_clock> cpy2grouped_parameters_start_point = std::chrono::system_clock::now();

    for (int i = 0; i < mf_info->user_group_num; i++){
        unsigned int group_params_size = mf_info->user_group_size[i] * mf_info->params.k; 
        sgd_info->user_group_ptr[i] = (__half*)malloc(sizeof(__half)*group_params_size);
    }
    
    for (int i = 0; i < mf_info->item_group_num; i++){
        unsigned int group_params_size = mf_info->item_group_size[i] * mf_info->params.k; 
        sgd_info->item_group_ptr[i] = (__half*)malloc(sizeof(__half)*group_params_size);
    }

    for (int i = 0; i < mf_info->max_user; i++){
        unsigned int user_group = mf_info->user_index_info[i].g;
        unsigned int index = mf_info->user_index_info[i].v;

        for (int k = 0; k < mf_info->params.k; k++){
            ((__half*)sgd_info->user_group_ptr[user_group])[index * mf_info->params.k + k] = __float2half_rn(sgd_info->p[i * mf_info->params.k + k]);
        }
    }

    for (int i = 0; i < mf_info->max_item; i++){
        unsigned int item_group = mf_info->item_index_info[i].g;
        unsigned int index = mf_info->item_index_info[i].v;

        for (int k = 0; k < mf_info->params.k; k++){
            ((__half*)sgd_info->item_group_ptr[item_group])[index * mf_info->params.k + k] = __float2half_rn(sgd_info->q[i * mf_info->params.k + k]);
        }
    }

    //* TRANSFER GROUPED PARAMETER TO GPU
    for (int i = 0; i < mf_info->user_group_num; i++){
        unsigned int group_params_size = mf_info->user_group_size[i] * mf_info->params.k; 
        hipMalloc((void**)&sgd_info->user_group_d_ptr[i], sizeof(__half) * group_params_size);
        hipMemcpy(sgd_info->user_group_d_ptr[i], sgd_info->user_group_ptr[i], sizeof(__half) * group_params_size, hipMemcpyHostToDevice);
    }

    for (int i = 0; i < mf_info->item_group_num; i++){
        unsigned int group_params_size = mf_info->item_group_size[i] * mf_info->params.k; 
        hipMalloc((void**)&sgd_info->item_group_d_ptr[i], sizeof(__half) * group_params_size);
        hipMemcpy(sgd_info->item_group_d_ptr[i], sgd_info->item_group_ptr[i], sizeof(__half) * group_params_size, hipMemcpyHostToDevice);
    }
    cpy2grouped_parameters_exec_time = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - cpy2grouped_parameters_start_point).count();

    hipMemcpy(sgd_info->d_user_group_ptr, sgd_info->user_group_d_ptr, sizeof(void**) * mf_info->user_group_num, hipMemcpyHostToDevice);
    hipMemcpy(sgd_info->d_item_group_ptr, sgd_info->item_group_d_ptr, sizeof(void**) * mf_info->item_group_num, hipMemcpyHostToDevice);

    cout << "\n<User & item parameter copy exec time (micro sec)>" << endl;
    cout << "Copy parameters             : " << cpy2grouped_parameters_exec_time << endl; 
}

void cpy2grouped_parameters_gpu(Mf_info *mf_info, SGD *sgd_info){
    // double cpy2grouped_parameters_exec_time = 0;
    // double alloc_device_host_parameter_exec_time = 0;

    // std::chrono::time_point<std::chrono::system_clock> cpy2grouped_parameters_start_point = std::chrono::system_clock::now();
    for (int i = 0; i < mf_info->user_group_num; i++){
        unsigned int group_params_size = mf_info->user_group_size[i] * mf_info->params.k; 
        hipMalloc((void**)&sgd_info->user_group_d_ptr[i], sizeof(__half) * group_params_size);
        // sgd_info->user_group_ptr[i] = (__half*)malloc(sizeof(__half)*group_params_size);
        hipHostMalloc(&sgd_info->user_group_ptr[i], sizeof(__half)*group_params_size);
        // sgd_info->user_group_ptr[i] = malloc(sizeof(__half) * group_params_size);
    }

    for (int i = 0; i < mf_info->item_group_num; i++){
        unsigned int group_params_size = mf_info->item_group_size[i] * mf_info->params.k; 
        hipMalloc((void**)&sgd_info->item_group_d_ptr[i], sizeof(__half) * group_params_size);
        // sgd_info->item_group_ptr[i] = (__half*)malloc(sizeof(__half)*group_params_size);
        hipHostMalloc(&sgd_info->item_group_ptr[i], sizeof(__half)*group_params_size);
        // sgd_info->item_group_ptr[i] = malloc(sizeof(__half) * group_params_size);
    }

    // alloc_device_host_parameter_exec_time = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - cpy2grouped_parameters_start_point).count();
    // cpy2grouped_parameters_exec_time += alloc_device_host_parameter_exec_time;
    
    hipMemcpy(sgd_info->d_user_group_ptr, sgd_info->user_group_d_ptr, sizeof(void**) * mf_info->user_group_num, hipMemcpyHostToDevice);
    hipMemcpy(sgd_info->d_item_group_ptr, sgd_info->item_group_d_ptr, sizeof(void**) * mf_info->item_group_num, hipMemcpyHostToDevice);
    
    unsigned int w_num = 2048;
    unsigned int block_size = 256;
    // cpy2grouped_parameters_start_point = std::chrono::system_clock::now();
    
    cpyparams2grouped_params<<<(w_num)/(block_size/32), block_size>>>(sgd_info->d_p, (__half**)sgd_info->d_user_group_ptr, mf_info->d_user_index_info, mf_info->params.k, mf_info->max_user);
    cpyparams2grouped_params<<<(w_num)/(block_size/32), block_size>>>(sgd_info->d_q, (__half**)sgd_info->d_item_group_ptr, mf_info->d_item_index_info, mf_info->params.k, mf_info->max_item);
    hipDeviceSynchronize();
    gpuErr(hipPeekAtLastError());
    // cpy2grouped_parameters_exec_time = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - cpy2grouped_parameters_start_point).count();
    
    // cout << "\n<User & item parameter copy exec time (micro sec)>" << endl;
    // cout << "Alloc parameter host & device time : " << alloc_device_host_parameter_exec_time << endl;
    // cout << "Copy parameters                    : " << cpy2grouped_parameters_exec_time << endl; 
}

void cpy2grouped_parameters_gpu_float_version(Mf_info *mf_info, SGD *sgd_info){
    double cpy2grouped_parameters_exec_time = 0;
    double alloc_device_host_parameter_exec_time = 0;

    std::chrono::time_point<std::chrono::system_clock> cpy2grouped_parameters_start_point = std::chrono::system_clock::now();
    for (int i = 0; i < mf_info->user_group_num; i++){
        unsigned int group_params_size = mf_info->user_group_size[i] * mf_info->params.k; 
        hipMalloc((void**)&sgd_info->user_group_d_ptr[i], sizeof(float) * group_params_size);
        // sgd_info->user_group_ptr[i] = (__half*)malloc(sizeof(__half)*group_params_size);
        hipHostMalloc(&sgd_info->user_group_ptr[i], sizeof(float)*group_params_size);
        // sgd_info->user_group_ptr[i] = malloc(sizeof(__half) * group_params_size);
    }

    for (int i = 0; i < mf_info->item_group_num; i++){
        unsigned int group_params_size = mf_info->item_group_size[i] * mf_info->params.k; 
        hipMalloc((void**)&sgd_info->item_group_d_ptr[i], sizeof(float) * group_params_size);
        // sgd_info->item_group_ptr[i] = (__half*)malloc(sizeof(__half)*group_params_size);
        hipHostMalloc(&sgd_info->item_group_ptr[i], sizeof(float)*group_params_size);
        // sgd_info->item_group_ptr[i] = malloc(sizeof(__half) * group_params_size);
    }

    alloc_device_host_parameter_exec_time = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - cpy2grouped_parameters_start_point).count();
    cpy2grouped_parameters_exec_time += alloc_device_host_parameter_exec_time;
    
    hipMemcpy(sgd_info->d_user_group_ptr, sgd_info->user_group_d_ptr, sizeof(void**) * mf_info->user_group_num, hipMemcpyHostToDevice);
    hipMemcpy(sgd_info->d_item_group_ptr, sgd_info->item_group_d_ptr, sizeof(void**) * mf_info->item_group_num, hipMemcpyHostToDevice);
    
    unsigned int w_num = 2048;
    unsigned int block_size = 256;
    cpy2grouped_parameters_start_point = std::chrono::system_clock::now();
    
    cpyparams2grouped_params_fp32_version<<<(w_num)/(block_size/32), block_size>>>(sgd_info->d_p, (float**)sgd_info->d_user_group_ptr, mf_info->d_user_index_info, mf_info->params.k, mf_info->max_user);
    cpyparams2grouped_params_fp32_version<<<(w_num)/(block_size/32), block_size>>>(sgd_info->d_q, (float**)sgd_info->d_item_group_ptr, mf_info->d_item_index_info, mf_info->params.k, mf_info->max_item);
    hipDeviceSynchronize();
    gpuErr(hipPeekAtLastError());
    cpy2grouped_parameters_exec_time = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - cpy2grouped_parameters_start_point).count();
    
    cout << "\n<User & item parameter copy exec time (micro sec)>" << endl;
    cout << "Alloc parameter host & device time : " << alloc_device_host_parameter_exec_time << endl;
    cout << "Copy parameters                    : " << cpy2grouped_parameters_exec_time << endl; 
}

void cpy2grouped_parameters_gpu_for_comparison_indexing(Mf_info *mf_info, SGD *sgd_info){

    double cpy2grouped_parameters_exec_time = 0;
    
    unsigned int* d_user_group_idx;
    unsigned int* d_item_group_idx;

    hipMalloc(&d_user_group_idx, sizeof(unsigned int) * mf_info->max_user);
    hipMalloc(&d_item_group_idx, sizeof(unsigned int) * mf_info->max_item);

    hipMemcpy(d_user_group_idx, mf_info->user_group_idx, sizeof(unsigned int) * mf_info->max_user, hipMemcpyHostToDevice);
    hipMemcpy(d_item_group_idx, mf_info->item_group_idx, sizeof(unsigned int) * mf_info->max_item, hipMemcpyHostToDevice);

    for (int i = 0; i < mf_info->user_group_num; i++){
        unsigned int group_params_size = mf_info->user_group_size[i] * mf_info->params.k; 
        hipMalloc((void**)&sgd_info->user_group_d_ptr[i], sizeof(__half) * group_params_size);
        // sgd_info->user_group_ptr[i] = (__half*)malloc(sizeof(__half)*group_params_size);
        hipHostMalloc(&sgd_info->user_group_ptr[i], sizeof(__half)*group_params_size);
    }

    for (int i = 0; i < mf_info->item_group_num; i++){
        unsigned int group_params_size = mf_info->item_group_size[i] * mf_info->params.k; 
        hipMalloc((void**)&sgd_info->item_group_d_ptr[i], sizeof(__half) * group_params_size);
        // sgd_info->item_group_ptr[i] = (__half*)malloc(sizeof(__half)*group_params_size);
        hipHostMalloc(&sgd_info->item_group_ptr[i], sizeof(__half)*group_params_size);
    }

    hipMemcpy(sgd_info->d_user_group_ptr, sgd_info->user_group_d_ptr, sizeof(void**) * mf_info->user_group_num, hipMemcpyHostToDevice);
    hipMemcpy(sgd_info->d_item_group_ptr, sgd_info->item_group_d_ptr, sizeof(void**) * mf_info->item_group_num, hipMemcpyHostToDevice);
    gpuErr(hipPeekAtLastError());

    unsigned int w_num = 2048;
    unsigned int block_size = 256;
    // std::chrono::time_point<std::chrono::system_clock> cpy2grouped_parameters_start_point = std::chrono::system_clock::now();
    if (mf_info->params.k == 128){
        cpyparams2grouped_params_for_comparison_indexing<<<(w_num)/(block_size/32), block_size, sizeof(unsigned int)*(mf_info->user_group_num+1)>>>(sgd_info->d_p, (__half**)sgd_info->d_user_group_ptr, mf_info->d_user_group_end_idx, d_user_group_idx, mf_info->d_user2sorted_idx, mf_info->params.k, mf_info->max_user, mf_info->user_group_num);
        cpyparams2grouped_params_for_comparison_indexing<<<(w_num)/(block_size/32), block_size, sizeof(unsigned int)*(mf_info->item_group_num+1)>>>(sgd_info->d_q, (__half**)sgd_info->d_item_group_ptr, mf_info->d_item_group_end_idx, d_item_group_idx, mf_info->d_item2sorted_idx, mf_info->params.k, mf_info->max_item, mf_info->item_group_num);
    }else if (mf_info->params.k == 64){
        cpyparams2grouped_params_for_comparison_indexing_k64<<<(w_num)/(block_size/32), block_size, sizeof(unsigned int)*(mf_info->user_group_num+1)>>>(sgd_info->d_p, (__half**)sgd_info->d_user_group_ptr, mf_info->d_user_group_end_idx, d_user_group_idx, mf_info->d_user2sorted_idx, mf_info->params.k, mf_info->max_user, mf_info->user_group_num);
        cpyparams2grouped_params_for_comparison_indexing_k64<<<(w_num)/(block_size/32), block_size, sizeof(unsigned int)*(mf_info->item_group_num+1)>>>(sgd_info->d_q, (__half**)sgd_info->d_item_group_ptr, mf_info->d_item_group_end_idx, d_item_group_idx, mf_info->d_item2sorted_idx, mf_info->params.k, mf_info->max_item, mf_info->item_group_num);
    }

    // cpyparams2grouped_params<<<(w_num)/(block_size/32), block_size>>>(sgd_info->d_p, (__half**)sgd_info->d_user_group_ptr, mf_info->d_user_index_info, mf_info->params.k, mf_info->max_user);
    // cpyparams2grouped_params<<<(w_num)/(block_size/32), block_size>>>(sgd_info->d_q, (__half**)sgd_info->d_item_group_ptr, mf_info->d_item_index_info, mf_info->params.k, mf_info->max_item);
    hipDeviceSynchronize();
    gpuErr(hipPeekAtLastError());
    // cpy2grouped_parameters_exec_time = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - cpy2grouped_parameters_start_point).count();
    
    // cout << "\n<User & item parameter copy exec time (micro sec)>" << endl;
    // cout << "Copy parameters             : " << cpy2grouped_parameters_exec_time << endl;
}

void cpy2grouped_parameters_gpu_for_division_indexing(Mf_info *mf_info, SGD *sgd_info){

    double cpy2grouped_parameters_exec_time = 0;

    for (int i = 0; i < mf_info->user_group_num; i++){
        unsigned int group_params_size = mf_info->user_group_size[i] * mf_info->params.k; 
        hipMalloc((void**)&sgd_info->user_group_d_ptr[i], sizeof(__half) * group_params_size);
        // sgd_info->user_group_ptr[i] = (__half*)malloc(sizeof(__half)*group_params_size);
        hipHostMalloc(&sgd_info->user_group_ptr[i], sizeof(__half)*group_params_size);
    }

    for (int i = 0; i < mf_info->item_group_num; i++){
        unsigned int group_params_size = mf_info->item_group_size[i] * mf_info->params.k; 
        hipMalloc((void**)&sgd_info->item_group_d_ptr[i], sizeof(__half) * group_params_size);
        // sgd_info->item_group_ptr[i] = (__half*)malloc(sizeof(__half)*group_params_size);
        hipHostMalloc(&sgd_info->item_group_ptr[i], sizeof(__half)*group_params_size);
    }

    hipMemcpy(sgd_info->d_user_group_ptr, sgd_info->user_group_d_ptr, sizeof(void**) * mf_info->user_group_num, hipMemcpyHostToDevice);
    hipMemcpy(sgd_info->d_item_group_ptr, sgd_info->item_group_d_ptr, sizeof(void**) * mf_info->item_group_num, hipMemcpyHostToDevice);
    
    unsigned int w_num = 2048;
    unsigned int block_size = 256;
    std::chrono::time_point<std::chrono::system_clock> cpy2grouped_parameters_start_point = std::chrono::system_clock::now();
    cpyparams2grouped_params_for_division_indexing<<<(w_num)/(block_size/32), block_size>>>(sgd_info->d_p, (__half**)sgd_info->d_user_group_ptr, mf_info->d_user2sorted_idx, mf_info->params.k, mf_info->max_user, (unsigned int)ceil(mf_info->max_user/(float)mf_info->user_group_num));
    cpyparams2grouped_params_for_division_indexing<<<(w_num)/(block_size/32), block_size>>>(sgd_info->d_q, (__half**)sgd_info->d_item_group_ptr, mf_info->d_item2sorted_idx, mf_info->params.k, mf_info->max_item, (unsigned int)ceil(mf_info->max_item/(float)mf_info->item_group_num));

    // cpyparams2grouped_params<<<(w_num)/(block_size/32), block_size>>>(sgd_info->d_p, (__half**)sgd_info->d_user_group_ptr, mf_info->d_user_index_info, mf_info->params.k, mf_info->max_user);
    // cpyparams2grouped_params<<<(w_num)/(block_size/32), block_size>>>(sgd_info->d_q, (__half**)sgd_info->d_item_group_ptr, mf_info->d_item_index_info, mf_info->params.k, mf_info->max_item);
    hipDeviceSynchronize();
    gpuErr(hipPeekAtLastError());
    cpy2grouped_parameters_exec_time = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - cpy2grouped_parameters_start_point).count();
    
    cout << "\n<User & item parameter copy exec time (micro sec)>" << endl;
    cout << "Copy parameters             : " << cpy2grouped_parameters_exec_time << endl; 
}

void cpy2grouped_parameters_gpu_for_division_indexing_float_version(Mf_info *mf_info, SGD *sgd_info){

    double cpy2grouped_parameters_exec_time = 0;

    for (int i = 0; i < mf_info->user_group_num; i++){
        unsigned int group_params_size = mf_info->user_group_size[i] * mf_info->params.k; 
        hipMalloc((void**)&sgd_info->user_group_d_ptr[i], sizeof(float) * group_params_size);
        // sgd_info->user_group_ptr[i] = (__half*)malloc(sizeof(__half)*group_params_size);
        hipHostMalloc(&sgd_info->user_group_ptr[i], sizeof(float)*group_params_size);
    }

    for (int i = 0; i < mf_info->item_group_num; i++){
        unsigned int group_params_size = mf_info->item_group_size[i] * mf_info->params.k; 
        hipMalloc((void**)&sgd_info->item_group_d_ptr[i], sizeof(float) * group_params_size);
        // sgd_info->item_group_ptr[i] = (__half*)malloc(sizeof(__half)*group_params_size);
        hipHostMalloc(&sgd_info->item_group_ptr[i], sizeof(float)*group_params_size);
    }

    hipMemcpy(sgd_info->d_user_group_ptr, sgd_info->user_group_d_ptr, sizeof(void**) * mf_info->user_group_num, hipMemcpyHostToDevice);
    hipMemcpy(sgd_info->d_item_group_ptr, sgd_info->item_group_d_ptr, sizeof(void**) * mf_info->item_group_num, hipMemcpyHostToDevice);
    
    unsigned int w_num = 2048;
    unsigned int block_size = 256;
    std::chrono::time_point<std::chrono::system_clock> cpy2grouped_parameters_start_point = std::chrono::system_clock::now();
    cpyparams2grouped_params_for_division_indexing_fp32_version<<<(w_num)/(block_size/32), block_size>>>(sgd_info->d_p, (float**)sgd_info->d_user_group_ptr, mf_info->d_user2sorted_idx, mf_info->params.k, mf_info->max_user, (unsigned int)ceil(mf_info->max_user/(float)mf_info->user_group_num));
    cpyparams2grouped_params_for_division_indexing_fp32_version<<<(w_num)/(block_size/32), block_size>>>(sgd_info->d_q, (float**)sgd_info->d_item_group_ptr, mf_info->d_item2sorted_idx, mf_info->params.k, mf_info->max_item, (unsigned int)ceil(mf_info->max_item/(float)mf_info->item_group_num));

    // cpyparams2grouped_params<<<(w_num)/(block_size/32), block_size>>>(sgd_info->d_p, (__half**)sgd_info->d_user_group_ptr, mf_info->d_user_index_info, mf_info->params.k, mf_info->max_user);
    // cpyparams2grouped_params<<<(w_num)/(block_size/32), block_size>>>(sgd_info->d_q, (__half**)sgd_info->d_item_group_ptr, mf_info->d_item_index_info, mf_info->params.k, mf_info->max_item);
    hipDeviceSynchronize();
    gpuErr(hipPeekAtLastError());
    cpy2grouped_parameters_exec_time = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - cpy2grouped_parameters_start_point).count();
    
    cout << "\n<User & item parameter copy exec time (micro sec)>" << endl;
    cout << "Copy parameters             : " << cpy2grouped_parameters_exec_time << endl; 
}

void transform_feature_vector_half2float(short *half_feature, float *float_feature, unsigned int dim, unsigned int k){
    float *gpu_float_feature;
    half *gpu_half_feature;

    hipMalloc(&gpu_half_feature, sizeof(half)*dim*k);
    hipMalloc(&gpu_float_feature, sizeof(float)*dim*k);
    gpuErr(hipPeekAtLastError());

    hipMemcpy(gpu_half_feature, half_feature, sizeof(half)*dim*k, hipMemcpyHostToDevice);
    gpuErr(hipPeekAtLastError());

    transform_half2float<<<(dim*k+255)/256, 256>>>(gpu_float_feature, gpu_half_feature, dim*k);
    hipDeviceSynchronize();

    gpuErr(hipPeekAtLastError());
    
    hipMemcpy(float_feature, gpu_float_feature, sizeof(float)*dim*k, hipMemcpyDeviceToHost);
    gpuErr(hipPeekAtLastError());
    
    hipFree(gpu_float_feature);
    hipFree(gpu_half_feature);
    gpuErr(hipPeekAtLastError());
}

void conversion_features_half(short *feature_vec, float *feature_vec_from ,unsigned int dim, unsigned int k){
    __half* gpu_vec;
    float* gpu_from_vec;

    hipMalloc(&gpu_vec, sizeof(__half) * dim * k);
    hipMalloc(&gpu_from_vec, sizeof(float) * dim * k);
    hipMemcpy(gpu_from_vec, feature_vec_from, sizeof(float) * dim * k, hipMemcpyHostToDevice);
    gpuErr(hipPeekAtLastError());

    transform_float2half<<< (dim * k + 255) / 256, 256>>>(gpu_vec, gpu_from_vec, dim * k);
    hipDeviceSynchronize();
    gpuErr(hipPeekAtLastError());
        
    hipMemcpy(feature_vec, gpu_vec, sizeof(__half) * dim * k, hipMemcpyDeviceToHost);
    hipFree(gpu_vec);
    hipFree(gpu_from_vec);
}

void transition_params_half2float(Mf_info *mf_info, SGD *sgd_info){
    int num_groups = 10000;
    mem_cpy_fp16tofp32<<<num_groups, 512>>>(sgd_info->d_p, sgd_info->d_half_p, mf_info->params.k * mf_info->max_user);
    mem_cpy_fp16tofp32<<<num_groups, 512>>>(sgd_info->d_q, sgd_info->d_half_q, mf_info->params.k * mf_info->max_item);
    hipDeviceSynchronize();
    gpuErr(hipPeekAtLastError());
}